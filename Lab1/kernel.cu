#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>

//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include <iostream>
#include <numeric>
// includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
using namespace std;

__global__ void sum(long long* input)
{
	/*const int tid = threadIdx.x;
	auto step_size = 1;
	int number_of_threads = blockDim.x;
	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) 
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
		}
		step_size <<= 1; 
		number_of_threads >>= 1;
	}
	  __syncthreads();
	*/
	
	  if(threadIdx.x < 2048) input[threadIdx.x] += input[threadIdx.x+2048];
	  __syncthreads();
	  if(threadIdx.x < 1024) input[threadIdx.x] += input[threadIdx.x+1024];
	  __syncthreads();
	  if(threadIdx.x < 512) input[threadIdx.x] += input[threadIdx.x+512];
	  __syncthreads();	
	  if(threadIdx.x < 256) input[threadIdx.x] += input[threadIdx.x+256];
	  __syncthreads();
	  if(threadIdx.x < 128) input[threadIdx.x] += input[threadIdx.x+128];
	  __syncthreads();
		  if(threadIdx.x < 64) input[threadIdx.x] += input[threadIdx.x+64];
	  __syncthreads();
		  if(threadIdx.x < 32) input[threadIdx.x] += input[threadIdx.x+32];
	  __syncthreads();
		  if(threadIdx.x < 16) input[threadIdx.x] += input[threadIdx.x+16];
	  __syncthreads();
		  if(threadIdx.x < 8) input[threadIdx.x] += input[threadIdx.x+8];
	  __syncthreads();
		  if(threadIdx.x < 4) input[threadIdx.x] += input[threadIdx.x+4];
	  __syncthreads();
		  if(threadIdx.x < 2) input[threadIdx.x] += input[threadIdx.x+2];
	  __syncthreads();
		  if(threadIdx.x == 0) input[threadIdx.x] += input[threadIdx.x+1];
	  __syncthreads();
}

int main()
{

    tryAgain: // ��� �����
	
    srand(time(NULL));          //����� �������
    int i,n;                    //��� �����
	long long *h;
	printf("Input array size: ");
    scanf("%d",&n);             //������ ������
    //int h[n];
	h = (long long*)malloc(n * sizeof(long long));
    
    for(i=0;i<n;i++)            //�������� ��������
	{
		h[i]=rand()%1699999+1699995;
		// cout << " " << h[i] << endl;
	}
        
	const auto count = n;
	const long long size = count * sizeof(long long);

	long long* d;

    auto elapsedTimeInMsGPU = 0.0f;
	float elapsedTimeInMsCPU = 0.0f;
	StopWatchInterface *timer = NULL;

	//GPU restart
	hipDeviceReset();

	//Entry point to mesure time
	hipEvent_t start, stop;
	//GPU timer
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	//SDK timer
	sdkCreateTimer(&timer);

	//start timer
	checkCudaErrors(hipEventRecord(start, 0));
	sdkStartTimer(&timer);	

	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count / 2 >>>(d);

	long long result;
	hipMemcpy(&result, d, sizeof(long long), hipMemcpyDeviceToHost);
	
	//Stop the timer
	checkCudaErrors(hipEventRecord(stop, 0));
		sdkStopTimer(&timer);
		elapsedTimeInMsCPU = sdkGetTimerValue(&timer);
	

	// make sure GPU has finished copying
	checkCudaErrors(hipDeviceSynchronize());

	//Finish point to mesure time
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMsGPU, start, stop));
	
	printf("Execution time in ms via GPU timer %f\n", elapsedTimeInMsGPU);

	cout << "Sum(GPU) is " << result << endl;

	
	result = 0;
	for (int i = 0; i < count; i++)
		result += h[i];

	printf("Execution time in ms via CPU timer %f\n", elapsedTimeInMsCPU);

	cout << "Sum(CPU) is " << result << endl;

	
	getchar();

	hipFree(d);
	delete[] h;

	goto tryAgain; // � ��� �������� goto
	
	return 0;
}