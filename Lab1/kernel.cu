#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>

//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include <iostream>
#include <numeric>
// includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	auto step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
		}

		step_size <<= 1; 
		number_of_threads >>= 1;
	}
	  __syncthreads();
}

int main()
{

    tryAgain: // ��� �����
	
    srand(time(NULL));          //����� �������
    int i,n;                    //��� �����
	int *h;
	printf("Input array size: ");
    scanf("%d",&n);             //������ ������
    //int h[n];
	h = (int*)malloc(n * sizeof(int));
    
    for(i=0;i<n;i++)            //�������� ��������
	{
		h[i]=rand()%1699999+1699995;
		// cout << " " << h[i] << endl;
	}
        

	const auto count = n;
	const int size = count * sizeof(int);

	int* d;

    auto elapsedTimeInMsGPU = 0.0f;
	float elapsedTimeInMsCPU = 0.0f;
	StopWatchInterface *timer = NULL;

	//GPU restart
	hipDeviceReset();

	//Entry point to mesure time
	hipEvent_t start, stop;
	//GPU timer
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	//SDK timer
	sdkCreateTimer(&timer);

	//start timer
	checkCudaErrors(hipEventRecord(start, 0));
	sdkStartTimer(&timer);	

	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count / 2 >>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
	
	//Stop the timer
	checkCudaErrors(hipEventRecord(stop, 0));
		sdkStopTimer(&timer);
		elapsedTimeInMsCPU = sdkGetTimerValue(&timer);
	

	// make sure GPU has finished copying
	checkCudaErrors(hipDeviceSynchronize());

	//Finish point to mesure time
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMsGPU, start, stop));
	
	printf("Execution time in ms via GPU timer %f\n", elapsedTimeInMsGPU);

	cout << "Sum(GPU) is " << result << endl;

	result = 0;
	for (int i = 0; i < count; i++)
		result= result+h[i];

	printf("Execution time in ms via CPU timer %f\n", elapsedTimeInMsCPU);

	cout << "Sum(CPU) is " << result << endl;

	
	getchar();

	hipFree(d);
	delete[] h;

	goto tryAgain; // � ��� �������� goto
	
	return 0;
}